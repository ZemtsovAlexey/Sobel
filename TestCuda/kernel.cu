
#include "hip/hip_runtime.h"



#include <stdio.h>
#include <math.h>

#include <assert.h>
#include <cstdlib>

struct Test
{
	double** Cord;
};

__global__ void kernel(int a, int b, int *c)
{
	*c = (a + b)*(a + b);
}

__global__ void VecAdd(const double* A, const double* B, double *C, int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N)
		*C += A[i] * B[i];
}

__global__ void SumMatrix(double *A, double* B, int length0, int length1, int length2)
{
	/*const int N = 30000;
	int idx = blockDim.x * blockIdx.x + threadIdx.x;*/
	const int blockSize = length1 * length2;
	//printf(" %d \n", 2);

	//printf("blockSize \r\n");

	//if (idx < N) {
		for (int i = 0; i < length0; i++) {
			for (int y = 0; y < length1; y++) {
				for (int x = 0; x < length2; x++) {
					//*A[y * length2 + x] += B[(i * blockSize) + (y * length2 + x)];
					A[y * length2 + x] += B[(i * blockSize) + (y * length2 + x)];
				}
			}
		}
	//}
}

__global__ void matrixMultiplicationKernel(double* A, double* B, double* C, int N) {

	int ROW = blockIdx.y*blockDim.y + threadIdx.y;
	int COL = blockIdx.x*blockDim.x + threadIdx.x;

	float tmpSum = 0;

	if (ROW < N && COL < N) {
		C[ROW * N + COL] = A[ROW * N + COL] + B[ROW * N + COL];

		/*char rowText[] = "row - X\n";
		rowText[6] = ROW + '0';
		printf(rowText);

		char colText[] = "col - X\n";
		colText[6] = COL + '0';
		printf(colText);

		char valText[] = "val - X\n";
		valText[6] = A[ROW * N + COL] + '0';
		printf(valText);*/
		// each thread computes one element of the block sub-matrix
		/*for (int i = 0; i < N; i++) {
		tmpSum += A[ROW * N + i] * B[i * N + COL];
		}*/
	}
	//C[ROW * N + COL] = tmpSum;
}

__global__ void matrixSum(double* A, double* C, int X, int Y, int Z) {

	int ROW = blockIdx.y*blockDim.y + threadIdx.y;
	int COL = blockIdx.x*blockDim.x + threadIdx.x;

	//float tmpSum = 0;

	if (ROW < Y && COL < X) {
		for (int i = 0; i < Z; i++) {
			C[ROW * X + COL] += A[(ROW * Y * X) + (ROW * X + COL)];
		}
		//tmpSum += A[ROW * N + COL];
	}
	//C[ROW * X + COL] = tmpSum;
}

__global__ void add_threads(double *a, double *b, int z) {

	/* threadIdx.x gives the thread ID in each block */

	/*char rowText[] = "row - X\n";
	rowText[6] = blockIdx.x + '0';
	printf(rowText);*/

	for (unsigned int i = 0; i < z; i++)
		b[blockIdx.x] += (2 / (1 + exp(-2 * a[blockIdx.x]))) - 1;

}

#define N 1000

__global__
void add(int *a, int *b) {
	int i = blockIdx.x;
	if (i<N) {
		b[i] = 2 * a[i];
	}
}

int main()
{
	////
	//// Create int arrays on the CPU.
	//// ('h' stands for "host".)
	////
	//int ha[N], hb[N];

	////
	//// Create corresponding int arrays on the GPU.
	//// ('d' stands for "device".)
	////
	//int *da, *db;
	//cudaMalloc((void **)&da, N * sizeof(int));
	//cudaMalloc((void **)&db, N * sizeof(int));

	////
	//// Initialise the input data on the CPU.
	////
	//for (int i = 0; i<N; ++i) {
	//	ha[i] = i;
	//}

	////
	//// Copy input data to array on GPU.
	////
	//cudaMemcpy(da, ha, N * sizeof(int), cudaMemcpyHostToDevice);

	////
	//// Launch GPU code with N threads, one per
	//// array element.
	////
	//add<<<N, 1 >>>(da, db);

	////
	//// Copy output array from GPU back to CPU.
	////
	//cudaMemcpy(hb, db, N * sizeof(int), cudaMemcpyDeviceToHost);

	//for (int i = 0; i<N; ++i) {
	//	printf("%d\n", hb[i]);
	//}

	////
	//// Free up the arrays on the GPU.
	////
	//cudaFree(da);
	//cudaFree(db);

	return 0;
}