#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include <stdio.h>
#include <stdlib.h>

struct GridStruct
{
	dim3 gridDim;
	dim3 threadsPerBlock;
};

__global__ void rot90(double* input, double* output, int width, int height)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (y < height && x < width)
		output[x * width + (height - 1 - y)] = input[y * width + x];
}

__global__ void rot180(double* input, double* output, int width, int height)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (y < height && x < width)
	{
		output[(height - 1 - y) * width + (width - 1 - x)] = input[y * width + x];
	}
}

__global__ void Multiply(double* result, double* a, double* b)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;

	result[x] = a[x] * b[x];
}

__global__ void Multiply2(double* result, double* input, double* weights, int len)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int i = x >= len ? x % len : x;

	if (i < len) {
		result[x] = input[i] * weights[x];
	}
}

__global__ void convolution(
	double* output,
	double* input,
	double* kernel,
	int inputWidth,
	int inputHeight,
	int kernelWidth,
	int kernelHeight,
	int outWidth,
	int outHeight
)
{
	//extern __shared__ double sh[];
	__shared__ double sum;

	int x = threadIdx.x + blockIdx.x;// *blockDim.x;
	int y = threadIdx.y + blockIdx.y;// *blockDim.y;

	if (x < inputWidth && y < inputHeight)
	{
		//double sum = 0;
		/*for (int ky = 0; ky < kernelHeight; ky++)
		{
			for (int kx = 0; kx < kernelHeight; kx++)
			{
				sum += (input[(y + ky) * inputWidth + (x + kx)] * kernel[ky * kernelWidth + kx]);
			}
		}*/

		sum = sum + (input[(y)* inputWidth + (x)] * kernel[threadIdx.y * kernelWidth + threadIdx.x]);

		__syncthreads();

		output[blockIdx.y * outWidth + blockIdx.x] = sum;
		sum = 0;
	}
}

//__global__ void ArraySum(float *array)
//
//{
//
//	int index = threadIdx.x;
//
//	sum = sum + array[index];
//
//	__syncthreads();
//
//}

__global__ void calcSum(
	double output, 
	double* input, 
	double* kernel, 
	int inputWidth, 
	int inputHeight,
	int kernelWidth,
	int kernelHeight
)
{
	extern __shared__ double sum;

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < inputWidth && y < inputHeight)
	{
		sum += (input[(y)* inputWidth + (x)] * kernel[threadIdx.y * kernelWidth + threadIdx.x]);

		__syncthreads();

		output = sum;
	}
}

//__global__
//void transpose_2(double* a, double* b, int N)
//{
//	__shared__ double sh[8][8];
//
//	int x = blockIdx.x * blockDim.x;
//	int y = blockIdx.y * blockDim.y;
//	int i = x + threadIdx.x;
//	int j = y + threadIdx.y;
//
//	sh[threadIdx.y][threadIdx.x] = a[j * N + i];
//
//	__syncthreads();
//
//	b[((x + threadIdx.x)) * N + (N - 1 - (y + threadIdx.y))] = sh[threadIdx.y][threadIdx.x];
//}

GridStruct getGridModel(int width, int length)
{
	GridStruct result;

	int height = length / width;
	int BSX = width > 127 ? 127 : width;
	int BSY = height > 127 ? 127 : height;

	int vx = width % BSX > 0 ? (width / BSX) + 1 : width / BSX;
	int vy = height % BSY > 0 ? (height / BSY) + 1 : height / BSY;

	result.gridDim = dim3(vx, vy, 1);
	result.threadsPerBlock = dim3(BSX, BSY, 1);

	return result;
}

extern "C" __declspec(dllexport) void Rot90GPU(double output[], double input[], int width, int length)
{
	double *dev_a, *dev_b;
	int height = length / width;

	hipMalloc(&dev_a, sizeof(double) * width * height);
	hipMalloc(&dev_b, sizeof(double) * width * height);
	hipMemcpy(dev_a, input, sizeof(double) * width * height, hipMemcpyHostToDevice);

	GridStruct grid = getGridModel(width, length);

	rot90<<<grid.gridDim, grid.threadsPerBlock>>>(dev_a, dev_b, width, height);

	hipDeviceSynchronize();

	hipMemcpy(output, dev_b, sizeof(double) * width * height, hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
}

extern "C" __declspec(dllexport) void Rot180GPU(double output[], double input[], int width, int length)
{
	double *dev_a, *dev_b;
	int height = length / width;

	hipHostMalloc(&dev_a, sizeof(double) * width * height);
	hipHostMalloc(&dev_b, sizeof(double) * width * height);
	hipMemcpy(dev_a, input, sizeof(double) * width * height, hipMemcpyHostToHost);

	GridStruct grid = getGridModel(width, length);

	rot180 << <grid.gridDim, grid.threadsPerBlock >> > (dev_a, dev_b, width, height);

	hipDeviceSynchronize();

	hipMemcpy(output, dev_b, sizeof(double) * width * height, hipMemcpyHostToHost);

	hipFree(dev_a);
	hipFree(dev_b);
}

extern "C" __declspec(dllexport) void MultiplyGPU(double output[], double input[], double weights[], int len)
{
	double *dev_output, *dev_a, *dev_b;

	hipMalloc(&dev_output, sizeof(double) * len);
	hipMalloc(&dev_a, sizeof(double) * len);
	hipMalloc(&dev_b, sizeof(double) * len);
	hipMemcpy(dev_a, input, sizeof(double) * len, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, weights, sizeof(double) * len, hipMemcpyHostToDevice);

	const int size = 511;
	int threads = len > size ? size : len;
	int blocks = len % size > 0 ? len / threads + 1 : len / threads;

	Multiply << <blocks, threads >> > (dev_output, dev_a, dev_b);

	hipDeviceSynchronize();

	hipMemcpy(output, dev_output, sizeof(double) * len, hipMemcpyDeviceToHost);

	hipFree(dev_output);
	hipFree(dev_a);
	hipFree(dev_b);
}

extern "C" __declspec(dllexport) void Multiply2GPU(double output[], double input[], double weights[], int len, int wlen, int nlen)
{
	double *dev_output, *dev_input, *dev_weights;

	hipMalloc(&dev_output, sizeof(double) * wlen);
	hipMalloc(&dev_input, sizeof(double) * len);
	hipMalloc(&dev_weights, sizeof(double) * wlen);
	hipMemcpy(dev_input, input, sizeof(double) * len, hipMemcpyHostToDevice);
	hipMemcpy(dev_weights, weights, sizeof(double) * wlen, hipMemcpyHostToDevice);

	const int size = 500;
	int threads = wlen > size ? size : wlen;
	int blocks = wlen % size > 0 ? wlen / threads + 1 : wlen / threads;

	Multiply2 << <blocks, threads >> > (dev_output, dev_input, dev_weights, len);

	hipDeviceSynchronize();

	double *o = new double[wlen];

	hipMemcpy(o, dev_output, sizeof(double) * wlen, hipMemcpyDeviceToHost);

	for (int i = 0; i < wlen; i++) 
	{
		output[i / wlen] += *(o + i);
	}

	hipFree(dev_output);
	hipFree(dev_input);
	hipFree(dev_weights);
}

extern "C" __declspec(dllexport) void ConvolutionGPU(
	double output[],
	double input[],
	double kernel[],
	int inputWidth,
	int inputHeight,
	int kernelWidth,
	int kernelHeight,
	int outWidth,
	int outHeight
)
{
	double *dev_input, *dev_kernel, *dev_output;
	
	int BSX = kernelWidth;// > 512 ? 512 : kernelWidth;
	int BSY = kernelHeight;// > 512 ? 512 : kernelHeight;

	/*int vx = inputWidth % BSX > 0 ? (inputWidth / BSX) + 1 : inputWidth / BSX;
	int vy = inputHeight % BSY > 0 ? (inputHeight / BSY) + 1 : inputHeight / BSY;*/

	dim3 gridDim = dim3(outWidth, outHeight, 1);
	dim3 threadsPerBlock = dim3(kernelWidth, kernelHeight, 1);

	hipMalloc(&dev_input, sizeof(double) * inputWidth * inputHeight);
	hipMalloc(&dev_kernel, sizeof(double) * kernelWidth * kernelHeight);
	hipMalloc(&dev_output, sizeof(double) * outWidth * outHeight);
	hipMemcpy(dev_input, input, sizeof(double) * inputWidth * inputHeight, hipMemcpyHostToDevice);
	hipMemcpy(dev_kernel, kernel, sizeof(double) * kernelWidth * kernelHeight, hipMemcpyHostToDevice);

	convolution<<<gridDim, threadsPerBlock>>>(dev_output, dev_input, dev_kernel, inputWidth, inputHeight, kernelWidth, kernelHeight, outWidth, outHeight);

	hipDeviceSynchronize();

	hipMemcpy(output, dev_output, sizeof(double) * outWidth * outHeight, hipMemcpyDeviceToHost);

	hipFree(dev_input);
	hipFree(dev_kernel);
	hipFree(dev_output);
}