#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <chrono>
//#include <iostream>
//void addWithCuda(int *c, const int *a, const int *b, unsigned int size);
__global__ void addKernel(int *c, const int *a, const int *b)
{
	//c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];

	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	const int n = sizeof c;

	while (tid < n)
	{
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
}

__global__ void add_threads(double *C, double* A, int inLength, int outLength) {

	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	//const int n = sizeof Z * X * Y;
	//unsigned int rid = tid % (X * Y);

	/*char colText[] = "col - X\n";
	colText[6] = rid + '0';
	printf(colText);*/

	if (tid < inLength)
	{
		int rid = tid % outLength;


		char colText[] = "col - X Y\n";
		colText[6] = C[tid] + '0';
		colText[8] = A[rid] + '0';
		printf(colText);

		A[rid] = A[rid] + C[tid];

	}

	/*if (tid < inLength)
	{
		double sum = 0;
		int rid = tid % outLength;
		
		A[rid] += C[tid];
	}*/

}

__global__
void add(double *x, double *y, int n, int o)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	
	for (int i = index; i < n; i += stride)
	{
		y[i % o] = y[i % o] + x[i];
	}
}

__global__ void matrixSumKernel(double *A, const double *C, int X, int Y, int Z) {

	int ROW = blockIdx.y*blockDim.y + threadIdx.y;
	int COL = blockIdx.x*blockDim.x + threadIdx.x;
	int z = blockIdx.z*blockDim.z + threadIdx.z;

	A[ROW * X + COL] += C[(z * X * Y) + (ROW * X + COL)];

	/*if (ROW < Y && COL < X) {
		for (int i = 0; i < Z; i++) {
			A[ROW * X + COL] += C[(i * X * Y) + (ROW * X + COL)];
		}
	}*/
}
//const int N = 16;
//__global__ void MatAdd(float A[N][N], float B[N][N], float C[N][N]) {
//	int i = blockIdx.x * blockDim.x + threadIdx.x;
//	int j = blockIdx.y * blockDim.y + threadIdx.y;
//
//	if (i < N && j < N)
//		C[i][j] = A[i][j] + B[i][j];
//}

// Helper function for using CUDA to add vectors in parallel.
extern "C" __declspec(dllexport) void addWithCuda(int *c, const int *a, const int *b, unsigned int N)
{
	//Device array
	int *dev_a, *dev_b, *dev_c;

	//Allocate the memory on the GPU
	hipMalloc((void **)&dev_a, N * sizeof(int));
	hipMalloc((void **)&dev_b, N * sizeof(int));
	hipMalloc((void **)&dev_c, N * sizeof(int));

	//Copy Host array to Device array
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

	//Make a call to GPU kernel
	//addKernel <<< (N + 511) / 512, 512 >>> (dev_a, dev_b, dev_c);
	addKernel << < (N + 511) / 512, 512 >> > (dev_c, dev_a, dev_b);

	//Copy back to Host array from Device array
	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

	//Display the result
	/*for (int i = 0; i<N; i++)
	printf("%d + %d = %d\n", a[i], b[i], c[i]);*/

	//Free the Device array memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

extern "C" __declspec(dllexport) void matrixSum(double b[], const double a[], int x, int y, int z)
{
	//Device array
	double *dev_a, *dev_b;
	unsigned int N = z * y * x;
	int outLength = y * x;

	//Allocate the memory on the GPU
	hipMalloc((void **)&dev_a, z * x * y * sizeof(double));
	hipMalloc((void **)&dev_b, x * y * sizeof(double));

	//Copy Host array to Device array
	hipMemcpy(dev_a, a, z * x * y * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, x * y * sizeof(double), hipMemcpyHostToDevice);

	//Make a call to GPU kernel
	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	add << < numBlocks, blockSize >> > (dev_a, dev_b, N, outLength);
	//MatAdd << < numBlocks, threadsPerBlock >> > (a, b,);

	hipDeviceSynchronize();

	//Copy back to Host array from Device array
	hipMemcpy(b, dev_b, x * y * sizeof(double), hipMemcpyDeviceToHost);
	
	//Free the Device array memory
	hipFree(dev_a);
	hipFree(dev_b);
}
