#include "hip/hip_runtime.h"
/*
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}*/

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "sm_61_intrinsics.h"
#include "sm_60_atomic_functions.h"
#include "device_atomic_functions.h"
#include "device_double_functions.h"
#include "hip/device_functions.h"

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;

	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
		// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
	} while (assumed != old);

	//printf("%f\r\n", old);

	return __longlong_as_double(old);
}
#endif

struct GridStruct
{
	dim3 gridDim;
	dim3 threadsPerBlock;
};

__global__ void rot90(double* input, double* output, int width, int height)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (y < height && x < width)
		output[x * width + (height - 1 - y)] = input[y * width + x];
}

template <typename T>
__global__ void rot180(const T* input, T* output, int width, int height)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (y < height && x < width)
	{
		output[(height - 1 - y) * width + (width - 1 - x)] = input[y * width + x];
	}
}

__global__ void Multiply(double* result, double* a, double* b)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;

	result[x] = a[x] * b[x];
}

__global__ void Multiply2(float* result, float* input, float* weights, int len)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int i = x >= len ? x % len : x;

	result[x] = input[i] * weights[x];
}

__global__ void Multiply3(float* result, float* input, float* weights, int len, int neurons)
{
	int x = threadIdx.x;// +blockIdx.x * blockDim.x;

	if (x < neurons)
	{
		for (int i = 0; i < len; i++)
		{
			int n = neurons * x + i;
			result[x] += input[i] * weights[i];
		}
	}
}

const unsigned int MAX_FILTER_SIZE = 79;
__device__ __constant__ float d_cFilterKernel[MAX_FILTER_SIZE * MAX_FILTER_SIZE];

__global__ void backConvolution(
	float* output,
	float* input,
	float* kernel,
	int inputWidth,
	int inputHeight,
	int kernelWidth,
	int kernelHeight,
	int outWidth,
	int outHeight
)
{
	int step = 1;
	int padY = kernelHeight - step;
	int padX = kernelWidth - step;
	
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	
	x -= padX;
	y -= padY;
	//int x = blockIdx.x - padX;
	//int y = blockIdx.y - padY;

	for (int h = y < 0 ? 0 - y : 0; h < (y + kernelHeight > inputHeight ? (inputHeight - (y + kernelHeight)) + kernelHeight : kernelHeight); h++)
		for (int w = x < 0 ? 0 - x : 0; w < (x + kernelWidth > inputWidth ? (inputWidth - (x + kernelWidth)) + kernelWidth : kernelWidth); w++)
			output[(y + padY) * outWidth + (x + padX)] += input[((y + h) * inputWidth + x + w)] * kernel[h * kernelWidth + w];

		/*for (int y = -(padY); y < inputWidth + padY - step; y++)
		{
			for (int x = -(padX); x < inputHeight + padX - step; x++)
			{
				for (int h = y < 0 ? 0 - y : 0; h < (y + kernelHeight > inputHeight ? (inputHeight - (y + kernelHeight)) + kernelHeight : kernelHeight); h++)
					for (int w = x < 0 ? 0 - x : 0; w < (x + kernelWidth > inputWidth ? (inputWidth - (x + kernelWidth)) + kernelWidth : kernelWidth); w++)
						output[(y + padY) * outWidth + (x + padX)] += input[((y + h) * inputWidth + x + w)] * kernel[h * kernelWidth + w];
			}
		}*/
}

__global__ void convolution(
	float* output,
	float* input,
	float* kernel,
	int inputWidth,
	int inputHeight,
	int kernelWidth,
	int kernelHeight,
	int outWidth,
	int outHeight
)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x < inputWidth && y < inputHeight)
	{
		float sum = 0;

		for (int ky = 0; ky < kernelHeight; ky++)
		{
			for (int kx = 0; kx < kernelHeight; kx++)
			{
				output[y * outWidth + x] += (input[(y + ky) * inputWidth + (x + kx)] * kernel[ky * kernelWidth + kx]);
			}
		}
	}
}

template <typename T>
__global__ void imageFilteringKernel(const T* d_f, const unsigned int paddedW, const unsigned int paddedH, 	const int S, T* d_h, const unsigned int W, const unsigned int H)
{
	// Set the padding size and filter size
	unsigned int paddingSize = S;
	unsigned int filterSize = 2 * S + 1;

	// Set the pixel coordinate
	const unsigned int j = blockIdx.x * blockDim.x + threadIdx.x + paddingSize;
	const unsigned int i = blockIdx.y * blockDim.y + threadIdx.y + paddingSize;

	// The multiply-add operation for the pixel coordinate ( j, i )
	if (j >= paddingSize && j < paddedW - paddingSize && i >= paddingSize && i < paddedH - paddingSize) 
	{
		unsigned int oPixelPos = (i - paddingSize) * W + (j - paddingSize);
		d_h[oPixelPos] = 0.0;

		for (int k = -S; k <= S; k++) 
		{
			for (int l = -S; l <= S; l++) 
			{
				unsigned int iPixelPos = (i + k) * paddedW + (j + l);
				unsigned int coefPos = (k + S) * filterSize + (l + S);

				d_h[oPixelPos] += d_f[iPixelPos] * d_cFilterKernel[coefPos];
			}
		}
	}
}

template <typename T>
__global__ void imageFilteringKernelSh(const T* d_f, const unsigned int paddedW, const unsigned int paddedH, const unsigned int blockW, const unsigned int blockH, const int S,	T* d_h, const unsigned int W, const unsigned int H)
{

	//
	// Note that blockDim.(x,y) cannot be used instead of blockW and blockH,
	// because the size of a thread block is not equal to the size of a data block
	// due to the apron and the use of subblocks.
	//

	//
	// Set the size of a tile
	//
	const unsigned int tileW = blockW + 2 * S;
	const unsigned int tileH = blockH + 2 * S;

	// 
	// Set the number of subblocks in a tile
	//
	const unsigned int noSubBlocks = static_cast<unsigned int>(ceil(static_cast<double>(tileH) / static_cast<double>(blockDim.y)));

	//
	// Set the start position of a data block, which is determined by blockIdx. 
	// Note that since padding is applied to the input image, the origin of the block is ( S, S )
	//
	const unsigned int blockStartCol = blockIdx.x * blockW + S;
	const unsigned int blockEndCol = blockStartCol + blockW;

	const unsigned int blockStartRow = blockIdx.y * blockH + S;
	const unsigned int blockEndRow = blockStartRow + blockH;

	//
	// Set the position of the tile which includes the data block and its apron
	//
	const unsigned int tileStartCol = blockStartCol - S;
	const unsigned int tileEndCol = blockEndCol + S;
	const unsigned int tileEndClampedCol = min(tileEndCol, paddedW);

	const unsigned int tileStartRow = blockStartRow - S;
	const unsigned int tileEndRow = blockEndRow + S;
	const unsigned int tileEndClampedRow = min(tileEndRow, paddedH);

	//
	// Set the size of the filter kernel
	//
	const unsigned int kernelSize = 2 * S + 1;

	//
	// Shared memory for the tile
	//
	extern __shared__ T sData[];

	//
	// Copy the tile into shared memory
	//
	unsigned int tilePixelPosCol = threadIdx.x;
	unsigned int iPixelPosCol = tileStartCol + tilePixelPosCol;
	for (unsigned int subBlockNo = 0; subBlockNo < noSubBlocks; subBlockNo++) {

		unsigned int tilePixelPosRow = threadIdx.y + subBlockNo * blockDim.y;
		unsigned int iPixelPosRow = tileStartRow + tilePixelPosRow;

		if (iPixelPosCol < tileEndClampedCol && iPixelPosRow < tileEndClampedRow) { // Check if the pixel in the image
			unsigned int iPixelPos = iPixelPosRow * paddedW + iPixelPosCol;
			unsigned int tilePixelPos = tilePixelPosRow * tileW + tilePixelPosCol;
			sData[tilePixelPos] = d_f[iPixelPos];
		}

	}

	//
	// Wait for all the threads for data loading
	//
	__syncthreads();

	//
	// Perform convolution
	//
	tilePixelPosCol = threadIdx.x;
	iPixelPosCol = tileStartCol + tilePixelPosCol;
	for (unsigned int subBlockNo = 0; subBlockNo < noSubBlocks; subBlockNo++) {

		unsigned int tilePixelPosRow = threadIdx.y + subBlockNo * blockDim.y;
		unsigned int iPixelPosRow = tileStartRow + tilePixelPosRow;

		// Check if the pixel in the tile and image.
		// Note that the apron of the tile is excluded.
		if (iPixelPosCol >= tileStartCol + S && iPixelPosCol < tileEndClampedCol - S &&
			iPixelPosRow >= tileStartRow + S && iPixelPosRow < tileEndClampedRow - S) {

			// Compute the pixel position for the output image
			unsigned int oPixelPosCol = iPixelPosCol - S; // removing the origin
			unsigned int oPixelPosRow = iPixelPosRow - S;
			unsigned int oPixelPos = oPixelPosRow * W + oPixelPosCol;

			unsigned int tilePixelPos = tilePixelPosRow * tileW + tilePixelPosCol;

			d_h[oPixelPos] = 0.0;
			for (int i = -S; i <= S; i++) {
				for (int j = -S; j <= S; j++) {
					int tilePixelPosOffset = i * tileW + j;
					int coefPos = (i + S) * kernelSize + (j + S);
					d_h[oPixelPos] += sData[tilePixelPos + tilePixelPosOffset] * d_cFilterKernel[coefPos];
				}
			}

		}

	}

}

__global__ void sumArray(double* output, double* input, int outStride, int inStride)
{
	if (outStride > blockIdx.y && outStride > blockIdx.x)
	{
		for (int ky = 0; ky < inStride; ky++)
		{
			for (int kx = 0; kx < inStride; kx++)
			{
				output[blockIdx.y * outStride + blockIdx.x] += input[(blockIdx.y + ky) * inStride + (blockIdx.x + kx)];
			}
		}
	}
}

//__global__ void ArraySum(float *array)
//
//{
//
//	int index = threadIdx.x;
//
//	sum = sum + array[index];
//
//	__syncthreads();
//
//}

__global__ void calcSum(
	double output,
	double* input,
	double* kernel,
	int inputWidth,
	int inputHeight,
	int kernelWidth,
	int kernelHeight
)
{
	extern __shared__ double sum;

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < inputWidth && y < inputHeight)
	{
		sum += (input[(y)* inputWidth + (x)] * kernel[threadIdx.y * kernelWidth + threadIdx.x]);

		//__syncthreads();

		output = sum;
	}
}

//__global__
//void transpose_2(double* a, double* b, int N)
//{
//	__shared__ double sh[8][8];
//
//	int x = blockIdx.x * blockDim.x;
//	int y = blockIdx.y * blockDim.y;
//	int i = x + threadIdx.x;
//	int j = y + threadIdx.y;
//
//	sh[threadIdx.y][threadIdx.x] = a[j * N + i];
//
//	__syncthreads();
//
//	b[((x + threadIdx.x)) * N + (N - 1 - (y + threadIdx.y))] = sh[threadIdx.y][threadIdx.x];
//}

/*

__global__ void ArraySum(float* array)
{
	int index = threadIdx.x;

	sum = sum + array[index];

	__syncthreads();
}*/

GridStruct getGridModel(int width, int length)
{
	GridStruct result;

	int height = length / width;
	int BSX = width > 30 ? 30 : width;
	int BSY = height > 30 ? 30 : height;

	int vx = width % BSX > 0 ? (width / BSX) + 1 : width / BSX;
	int vy = height % BSY > 0 ? (height / BSY) + 1 : height / BSY;

	result.gridDim = dim3(vx, vy, 1);
	result.threadsPerBlock = dim3(BSX, BSY, 1);

	return result;
}

extern "C" __declspec(dllexport) void Rot90GPU(double output[], double input[], int width, int length)
{
	double* dev_a, * dev_b;
	int height = length / width;

	hipMalloc((void**)&dev_a, sizeof(double) * width * height);
	hipMalloc((void**)&dev_b, sizeof(double) * width * height);
	hipMemcpy(dev_a, input, sizeof(double) * width * height, hipMemcpyHostToDevice);

	GridStruct grid = getGridModel(width, length);

	rot90 <<<grid.gridDim, grid.threadsPerBlock>>> (dev_a, dev_b, width, height);

	hipDeviceSynchronize();

	hipMemcpy(output, dev_b, sizeof(double) * width * height, hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
}

extern "C" __declspec(dllexport) void Rot180GPU(float output[], float input[], int width, int length)
{
	float* dev_a, * dev_b;
	int height = length / width;

	hipMalloc((void**)&dev_a, sizeof(float) * length);
	hipMalloc((void**)&dev_b, sizeof(float) * length);
	hipMemcpy(dev_a, input, sizeof(float) * length, hipMemcpyHostToDevice);

	GridStruct grid = getGridModel(width, length);
	/*int BSX = width > 30 ? 30 : width;
	int BSY = height > 30 ? 30 : height;

	dim3 gridDim = dim3(width / BSX + 1, height / BSY + 1, 1);
	dim3 threadsPerBlock = dim3(BSX, BSY, 1);*/

	rot180 <<<grid.gridDim, grid.threadsPerBlock >>> (dev_a, dev_b, width, height);

	hipDeviceSynchronize();

	hipMemcpy(output, dev_b, sizeof(float) * length, hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
}

extern "C" __declspec(dllexport) void MultiplyGPU(double output[], double input[], double weights[], int len)
{
	double* dev_output, * dev_a, * dev_b;

	hipMalloc((void**)&dev_output, sizeof(double) * len);
	hipMalloc((void**)&dev_a, sizeof(double) * len);
	hipMalloc((void**)&dev_b, sizeof(double) * len);
	hipMemcpy(dev_a, input, sizeof(double) * len, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, weights, sizeof(double) * len, hipMemcpyHostToDevice);

	const int size = 511;
	int threads = len > size ? size : len;
	int blocks = len % size > 0 ? len / threads + 1 : len / threads;

	Multiply <<<blocks, threads>>> (dev_output, dev_a, dev_b);

	hipDeviceSynchronize();

	hipMemcpy(output, dev_output, sizeof(double) * len, hipMemcpyDeviceToHost);

	hipFree(dev_output);
	hipFree(dev_a);
	hipFree(dev_b);
}

extern "C" __declspec(dllexport) void Multiply2GPU(float output[], float input[], float weights[], int len, int wlen, int nlen)
{
	float* dev_output, * dev_input, * dev_weights;

	hipMalloc((void**)&dev_output, sizeof(float) * nlen);
	hipMalloc((void**)&dev_input, sizeof(float) * len);
	hipMalloc((void**)&dev_weights, sizeof(float) * wlen);
	hipMemcpy(dev_input, input, sizeof(float) * len, hipMemcpyHostToDevice);
	hipMemcpy(dev_weights, weights, sizeof(float) * wlen, hipMemcpyHostToDevice);

	const int size = 127;
	int threads = wlen > size ? size : wlen;
	int blocks = wlen % size > 0 ? wlen / threads + 1 : wlen / threads;

	Multiply3 <<<1, nlen>>> (dev_output, dev_input, dev_weights, len, nlen);

	hipDeviceSynchronize();

	hipMemcpy(output, dev_output, sizeof(float) * nlen, hipMemcpyDeviceToHost);

	/*for (int i = 0; i < wlen; i++)
	{
		output[i / len] += *(o + i);
	}*/

	hipFree(dev_output);
	hipFree(dev_input);
	hipFree(dev_weights);
}

extern "C" __declspec(dllexport) void ConvolutionGPU(
	float output[],
	float input[],
	float kernel[],
	int inputWidth,
	int inputHeight,
	int kernelWidth,
	int kernelHeight,
	int outWidth,
	int outHeight
)
{
	float* dev_input, * dev_output, * dev_kernel;

	int BSX = outWidth > 32 ? 32 : outWidth;
	int BSY = outWidth > 32 ? 32 : outWidth;

	dim3 gridDim = dim3(outWidth / BSX + 1, outWidth / BSY + 1, 1);
	dim3 threadsPerBlock = dim3(BSX, BSY, 1);

	hipMalloc((void**)&dev_input, sizeof(float) * inputWidth * inputHeight);
	hipMalloc((void**)&dev_output, sizeof(float) * outWidth * outHeight);
	hipMalloc((void**)&dev_kernel, sizeof(float) * kernelWidth * kernelHeight);

	hipMemcpy(dev_input, input, sizeof(float) * inputWidth * inputHeight, hipMemcpyHostToDevice);
	hipMemcpy(dev_kernel, kernel, sizeof(float) * kernelWidth * kernelHeight, hipMemcpyHostToDevice);

	convolution<<<gridDim, threadsPerBlock >>>(dev_output, dev_input, dev_kernel, inputWidth, inputHeight, kernelWidth, kernelHeight, outWidth, outHeight);

	hipDeviceSynchronize();

	hipMemcpy(output, dev_output, sizeof(float) * outWidth * outHeight, hipMemcpyDeviceToHost);

	hipFree(dev_input);
	hipFree(dev_output);
	hipFree(dev_kernel);
}

//int iDivUp(int a, int b) { return ((a % b) != 0) ? (a / b + 1) : (a / b); }
inline unsigned int iDivUp(const unsigned int& a, const unsigned int& b) { return (a % b != 0) ? (a / b + 1) : (a / b); }

extern "C" __declspec(dllexport) void ConvolutionGPU2(
	float output[],
	float input[],
	float kernel[],
	int inputWidth,
	int inputHeight,
	int kernelWidth,
	int kernelHeight,
	int outWidth,
	int outHeight
)
{
	float* dev_input, * dev_output;

	unsigned int filterKernelSizeByte = kernelWidth * kernelHeight * sizeof(float);
	hipMemcpyToSymbol(HIP_SYMBOL(d_cFilterKernel), kernel, filterKernelSizeByte, 0, hipMemcpyHostToDevice);

	const unsigned int S = (kernelWidth - 1) / 2;
	const dim3 grid(iDivUp(inputWidth, kernelWidth), iDivUp(inputHeight, kernelHeight));
	const dim3 threadBlock(kernelWidth, kernelHeight);

	hipMalloc((void**)& dev_input, sizeof(float) * inputWidth * inputHeight);
	hipMalloc((void**)& dev_output, sizeof(float) * outWidth * outHeight);
	hipMemcpy(dev_input, input, sizeof(float) * inputWidth * inputHeight, hipMemcpyHostToDevice);

	imageFilteringKernel <<<grid, threadBlock >>> (dev_input, inputWidth, inputHeight, S, dev_output, outWidth, outHeight);

	hipDeviceSynchronize();

	hipMemcpy(output, dev_output, sizeof(float) * outWidth * outHeight, hipMemcpyDeviceToHost);

	hipFree(dev_input);
	hipFree(dev_output);
	hipFree(d_cFilterKernel);
}

extern "C" __declspec(dllexport) void ConvolutionGPU3(
	float output[],
	float input[],
	float kernel[],
	int inputWidth,
	int inputHeight,
	int kernelWidth,
	int kernelHeight,
	int outWidth,
	int outHeight
)
{
	float* dev_input, * dev_output;

	unsigned int filterKernelSizeByte = kernelWidth * kernelHeight * sizeof(float);
	hipMemcpyToSymbol(HIP_SYMBOL(d_cFilterKernel), kernel, filterKernelSizeByte, 0, hipMemcpyHostToDevice);

	const unsigned int S = (kernelWidth - 1) / 2;

	const unsigned int blockW = 32 - kernelWidth;
	const unsigned int blockH = 32 - kernelHeight;
	/*const unsigned int tileW = blockW + 2 * S;
	const unsigned int tileH = blockH + 2 * S;
	const dim3 grid(iDivUp(S, blockW), iDivUp(S, blockH));
	const dim3 threadBlock(tileW, tileH);*/

	const unsigned int tileW = blockW + 2 * S;
	const unsigned int tileH = blockH + 2 * S;
	const dim3 grid(iDivUp(outWidth, blockW), iDivUp(outHeight, blockH));
	const dim3 threadBlock(tileW, tileH);

	const unsigned int sharedMemorySizeByte = tileW * tileH * sizeof(float);

	hipMalloc((void**)& dev_input, sizeof(float) * inputWidth * inputHeight);
	hipMalloc((void**)& dev_output, sizeof(float) * outWidth * outHeight);
	hipMemcpy(dev_input, input, sizeof(float) * inputWidth * inputHeight, hipMemcpyHostToDevice);

	imageFilteringKernelSh << <grid, threadBlock, sharedMemorySizeByte >> > (dev_input, inputWidth, inputHeight, blockH, blockW, S, dev_output, outWidth, outHeight);

	hipDeviceSynchronize();

	hipMemcpy(output, dev_output, sizeof(float) * outWidth * outHeight, hipMemcpyDeviceToHost);

	hipFree(dev_input);
	hipFree(dev_output);
	hipFree(d_cFilterKernel);
}

extern "C" __declspec(dllexport) void BackConvolutionGPU(
	float output[],
	float input[],
	float kernel[],
	int inputWidth,
	int inputHeight,
	int kernelWidth,
	int kernelHeight,
	int outWidth,
	int outHeight
)
{
	float* dev_input, * dev_output, * dev_kernel;

	int step = 1;
	int padY = kernelHeight - step;
	int padX = kernelWidth - step;

	dim3 gridDim = dim3(inputWidth + padY - step, inputHeight + padX - step, 1);
	dim3 threadsPerBlock = dim3(outWidth, outHeight, 1);

	hipMalloc((void**)& dev_input, sizeof(float) * inputWidth * inputHeight);
	hipMalloc((void**)& dev_output, sizeof(float) * outWidth * outHeight);
	hipMalloc((void**)& dev_kernel, sizeof(float) * kernelWidth * kernelHeight);

	hipMemcpy(dev_input, input, sizeof(float) * inputWidth * inputHeight, hipMemcpyHostToDevice);
	hipMemcpy(dev_kernel, kernel, sizeof(float) * kernelWidth * kernelHeight, hipMemcpyHostToDevice);

	backConvolution << <gridDim, threadsPerBlock >> > (dev_output, dev_input, dev_kernel, inputWidth, inputHeight, kernelWidth, kernelHeight, outWidth, outHeight);

	hipDeviceSynchronize();

	hipMemcpy(output, dev_output, sizeof(float) * outWidth * outHeight, hipMemcpyDeviceToHost);

	hipFree(dev_input);
	hipFree(dev_output);
	hipFree(dev_kernel);
}

extern "C" __declspec(dllexport) void BackConvolutionGPU2(
	float output[],
	float input[],
	float kernel[],
	int inputWidth,
	int inputHeight,
	int kernelWidth,
	int kernelHeight,
	int outWidth,
	int outHeight
)
{
	float* dev_input, * dev_output, * dev_kernel;

	int step = 1;
	int padY = kernelHeight - step;
	int padX = kernelWidth - step;

	/*dim3 gridDim = dim3(inputWidth + padY - step, inputHeight + padX - step, 1);
	dim3 threadsPerBlock = dim3(outWidth, outHeight, 1);*/

	int BSX = outWidth > 32 ? 32 : outWidth;
	int BSY = outWidth > 32 ? 32 : outWidth;

	dim3 gridDim = dim3(outWidth / BSX + 1, outWidth / BSY + 1, 1);
	dim3 threadsPerBlock = dim3(BSX, BSY, 1);

	hipMalloc((void**)& dev_input, sizeof(float) * inputWidth * inputHeight);
	hipMalloc((void**)& dev_output, sizeof(float) * outWidth * outHeight);
	hipMalloc((void**)& dev_kernel, sizeof(float) * kernelWidth * kernelHeight);

	hipMemcpy(dev_input, input, sizeof(float) * inputWidth * inputHeight, hipMemcpyHostToDevice);
	hipMemcpy(dev_kernel, kernel, sizeof(float) * kernelWidth * kernelHeight, hipMemcpyHostToDevice);

	GridStruct grid = getGridModel(kernelWidth, kernelWidth * kernelHeight);

	rot180 << <grid.gridDim, grid.threadsPerBlock >> > (dev_kernel, dev_kernel, kernelWidth, kernelWidth * kernelHeight);
	backConvolution << <gridDim, threadsPerBlock >> > (dev_output, dev_input, dev_kernel, inputWidth, inputHeight, kernelWidth, kernelHeight, outWidth, outHeight);

	hipDeviceSynchronize();

	hipMemcpy(output, dev_output, sizeof(float) * outWidth * outHeight, hipMemcpyDeviceToHost);

	hipFree(dev_input);
	hipFree(dev_output);
	hipFree(dev_kernel);
}